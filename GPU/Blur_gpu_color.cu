#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <cmath>
#include <sstream>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>

__device__ int *iToxy(int, int);
__device__ int xyToi(int, int, int);
__device__ uchar3 prom_punto(uchar3*, int, int, int, int);
__global__ void cudaBlur(uchar3*, uchar3*, int, int, int, int);
inline int _ConvertSMVer2Cores(int, int);

int main(int n, char* argv[])
{
	int THREADS = 0;
	int BLOCKS = 0;
	int total_threads;
	int cuda_err = hipSuccess;
	int img_size = 0;
	float radio = -1;
	cv::Mat src;

	std::cout << CV_LOAD_IMAGE_GRAYSCALE << std::endl;

	//	Prueva que los parametros esten completos
	if (n != 4) {
		printf("blur <ruta img> <kernel> <thread>\n");
		return 0;
	}
	//Determina las caracteristicas de la targeta
	hipSetDevice(0);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	int max_threads = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
	std::cout << "cores disponibles por multiprocesador: " << max_threads << std::endl;

	//Calculando bloques y threads por bloque
	std::stringstream ss;
	ss << argv[3];
	ss >> total_threads;
	
	BLOCKS = (total_threads / (max_threads * 2)) + 1;
	THREADS = total_threads / BLOCKS;

	//	Determina el radio del kernel
	ss.clear();
	ss << argv[2];
	ss >> radio;
	if (radio < 1) {
		std::cerr << " Radio incorrecto para el Kernel, debe ser mayor a 1\n";
		return -1;
	}
	std::cout << "Kernel radio: " << (int)floor(radio) << std::endl;

	//	Carga la imagen en host
	src = cv::imread(argv[1], CV_LOAD_IMAGE_COLOR);
	if (!src.data) {
		std::cerr << "Error al leer la imagen\n";
		return -1;
	}
	std::cout << "Imagen: " << src.cols << "x" << src.rows << std::endl;

	//	Determina el tama�o del bloque de memoria para la imagen
	img_size = src.cols*src.rows * sizeof(uchar3);
	std::cout << "Imagen: " << ((double)img_size) / 1e6 << " Mb." << std::endl;

	//	Reservar la memoria en device para imagen original
	uchar3 *src_d;
	cuda_err = hipMalloc(&src_d, img_size);
	if (cuda_err != hipSuccess) {
		std::cerr << "Error al reservar memoria para imagen src en GPU\n";
		return -1;
	}
	std::cout << "Memoria de imagen src reservada en device\n";

	//	Reservar la memoria en device para imagen de respuesta
	uchar3 *ans_d;
	cuda_err = hipMalloc(&ans_d, img_size);
	if (cuda_err != hipSuccess) {
		std::cerr << "Error al reservar memoria para imagen ans en GPU\n";
		return -1;
	}
	std::cout << "Memoria de imagen ans reservada en device\n";

	//	Copiar imagen original al puntero src en device
	cuda_err = hipMemcpy(src_d, src.data, img_size, hipMemcpyHostToDevice);
	if (cuda_err != hipSuccess) {
		std::cerr << "Error al copiar imagen src a GPU\n";
		return -1;
	}
	std::cout << "imagen original copiada en device\n";

	std::cout << "Ejecutando " << BLOCKS << " bloques de " << THREADS << " threads." << std::endl;
	//	llamar proceso de blur paralelo
	cudaBlur <<< BLOCKS, THREADS >>> (src_d, ans_d, src.cols, src.rows, THREADS*BLOCKS, (int)std::floor(radio));

	//	Copia la respuesta del apuntador ans_d a src, desde el device al host
	cuda_err = hipMemcpy(src.data, ans_d, img_size, hipMemcpyDeviceToHost);
	if (cuda_err != hipSuccess) {
		std::cerr << "Error al copiar la respuesta de GPU al host\n";
		return -1;
	}
	std::cout << "Respuesta copiada al host\n";

	//	Liberar memoria en device
	cuda_err = hipFree(src_d);
	if (cuda_err != hipSuccess) {
		std::cerr << "Error al liberar memoria src en GPU\n";
		return -1;
	}
	cuda_err = hipFree(ans_d);
	if (cuda_err != hipSuccess) {
		std::cerr << "Error al liberar memoria ans en GPU\n";
		return -1;
	}
	std::cout << "Memoria liberada en device\n";

	//	namedWindow("final");
	//	imshow("final", dst);

	//	namedWindow("initial");
	//	imshow("initial", src);
	imwrite("../thread_blur.jpg", src);

	//	cv::waitKey(0);

	return 0;
}

/**
* Recorre los puntos del vector de datos de la imagen haciendo el blur a cada uno de ellos
*/
__global__ void cudaBlur(uchar3 *src, uchar3 *ans, int cols, int rows, int n_threads, int radio) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = id; i < cols*rows; i += n_threads) {
		*(ans + i) = prom_punto(src, i, rows, cols, radio);
	}
	return;
}

/**
* Convierte i a una cordenada de la forma (x,y).
* Retorna un apuntador con 2 pociciones reservadas.
* En la primera almacena el valor de x
* En la segunda almacena el valor de y
*/
__device__ int *iToxy(int i, int cols) {
	int *ans;
	ans = (int*)malloc(2 * sizeof(int));
	*ans = i%cols;
	*(ans + 1) = i / cols;
	return ans;
}

/**
* convierte una cordenada (x,y) a un valor i para array
* Retorna un entero con el valor de i
*/
__device__ int xyToi(int x, int y, int cols) {
	return cols*y + x;
}

/**
* Halla la suma promediada de los pixeles vecinos en base a un kernel
* src*			Un apuntador a el vector de datos de la imagen
* pos:			El indice del pixel, el indice en base a un array unidimencional
* rows, cols:	dimenciones de la imagen que se esta procesando
* radio:		El radio del kernel para los pixeles vecinos
* Retorna un entero con el valor de i
*/
__device__ uchar3 prom_punto(uchar3* src, int pos, int rows, int cols, int radio) {
	float  sum_peso;
	float3 sum = {0,0,0};

	sum_peso = 0;

	int *ptr_aux = iToxy(pos, cols);
	int x = *ptr_aux;
	int y = *(ptr_aux + 1);
	free(ptr_aux);

	for (int k = -radio; k <= radio; k++) {
		for (int j = -radio; j <= radio; j++) {
			if ((x + k) >= 0 && (x + k) < cols &&
				(y + j) >= 0 && (y + j) < rows) {
				float peso = exp(-(k*k + j*j) / (float)(2 * radio*radio)) / (3.141592 * 2 * radio*radio);
				sum.x += peso * (*(src + xyToi(x + k, y + j, cols))).x;
				sum.y += peso * (*(src + xyToi(x + k, y + j, cols))).y;
				sum.z += peso * (*(src + xyToi(x + k, y + j, cols))).z;
				sum_peso += peso;
			}
		}
	}
	
	uchar3 ans;

	ans.x = (uchar)std::floor(sum.x / sum_peso);
	ans.y = (uchar)std::floor(sum.y / sum_peso);
	ans.z = (uchar)std::floor(sum.z / sum_peso);

	return ans;
}

/**
 * Funcion de "cuda_helper.h" localizada en samples/common para determinar el numero de cores por multiprocesador del device
 */
inline int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct
	{
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] =
	{
		{ 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
		{ 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
		{ 0x30, 192 }, // Kepler Generation (SM 3.0) GK10x class
		{ 0x32, 192 }, // Kepler Generation (SM 3.2) GK10x class
		{ 0x35, 192 }, // Kepler Generation (SM 3.5) GK11x class
		{ 0x37, 192 }, // Kepler Generation (SM 3.7) GK21x class
		{ 0x50, 128 }, // Maxwell Generation (SM 5.0) GM10x class
		{ 0x52, 128 }, // Maxwell Generation (SM 5.2) GM20x class
		{ 0x53, 128 }, // Maxwell Generation (SM 5.3) GM20x class
		{ 0x60, 64 }, // Pascal Generation (SM 6.0) GP100 class
		{ 0x61, 128 }, // Pascal Generation (SM 6.1) GP10x class
		{ 0x62, 128 }, // Pascal Generation (SM 6.2) GP10x class
		{ -1, -1 }
	};

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1)
	{
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
		{
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	// If we don't find the values, we default use the previous one to run properly
	printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[index - 1].Cores);
	return nGpuArchCoresPerSM[index - 1].Cores;
}